#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include "compat.cuh"
#include "utils.cuh"

#define THREADS 1024

template <typename scalar_t> struct Cosine {
  static inline __device__ scalar_t dot(const scalar_t *a, const scalar_t *b,
                                        size_t size) {
    scalar_t result = 0;
    for (ptrdiff_t i = 0; i < size; i++) {
      result += a[i] * b[i];
    }
    return result;
  }

  static inline __device__ scalar_t norm(const scalar_t *a, size_t size) {
    scalar_t result = 0;
    for (ptrdiff_t i = 0; i < size; i++) {
      result += a[i] * a[i];
    }
    return sqrt(result);
  }
};

template <typename scalar_t>
__global__ void
knn_kernel(const scalar_t *__restrict__ x, const scalar_t *__restrict__ y,
           const int64_t *__restrict__ batch_x,
           const int64_t *__restrict__ batch_y, scalar_t *__restrict__ dist,
           int64_t *__restrict__ row, int64_t *__restrict__ col, size_t k,
           size_t dim, bool cosine) {

  const ptrdiff_t batch_idx = blockIdx.x;
  const ptrdiff_t idx = threadIdx.x;

  const ptrdiff_t start_idx_x = batch_x[batch_idx];
  const ptrdiff_t end_idx_x = batch_x[batch_idx + 1];

  const ptrdiff_t start_idx_y = batch_y[batch_idx];
  const ptrdiff_t end_idx_y = batch_y[batch_idx + 1];

  for (ptrdiff_t n_y = start_idx_y + idx; n_y < end_idx_y; n_y += THREADS) {

    for (ptrdiff_t k_idx = 0; k_idx < k; k_idx++) {
      row[n_y * k + k_idx] = n_y;
    }

    for (ptrdiff_t n_x = start_idx_x; n_x < end_idx_x; n_x++) {

      scalar_t tmp_dist = 0;
      if (cosine) {
        tmp_dist =
            Cosine<scalar_t>::norm(x, dim) * Cosine<scalar_t>::norm(y, dim) -
            Cosine<scalar_t>::dot(x, y, dim);
      } else {
        for (ptrdiff_t d = 0; d < dim; d++) {
          tmp_dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
                      (x[n_x * dim + d] - y[n_y * dim + d]);
        }
      }

      for (ptrdiff_t k_idx_1 = 0; k_idx_1 < k; k_idx_1++) {
        if (dist[n_y * k + k_idx_1] > tmp_dist) {
          for (ptrdiff_t k_idx_2 = k - 1; k_idx_2 > k_idx_1; k_idx_2--) {
            dist[n_y * k + k_idx_2] = dist[n_y * k + k_idx_2 - 1];
            col[n_y * k + k_idx_2] = col[n_y * k + k_idx_2 - 1];
          }
          dist[n_y * k + k_idx_1] = tmp_dist;
          col[n_y * k + k_idx_1] = n_x;
          break;
        }
      }
    }
  }
}

at::Tensor knn_cuda(at::Tensor x, at::Tensor y, size_t k, at::Tensor batch_x,
                    at::Tensor batch_y, bool cosine) {
  hipSetDevice(x.get_device());
  auto batch_sizes = (int64_t *)malloc(sizeof(int64_t));
  hipMemcpy(batch_sizes, batch_x[-1].DATA_PTR<int64_t>(), sizeof(int64_t),
             hipMemcpyDeviceToHost);
  auto batch_size = batch_sizes[0] + 1;

  batch_x = degree(batch_x, batch_size);
  batch_x = at::cat({at::zeros(1, batch_x.options()), batch_x.cumsum(0)}, 0);
  batch_y = degree(batch_y, batch_size);
  batch_y = at::cat({at::zeros(1, batch_y.options()), batch_y.cumsum(0)}, 0);

  auto dist = at::full(y.size(0) * k, 1e38, y.options());
  auto row = at::empty(y.size(0) * k, batch_y.options());
  auto col = at::full(y.size(0) * k, -1, batch_y.options());

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "knn_kernel", [&] {
    knn_kernel<scalar_t><<<batch_size, THREADS>>>(
        x.DATA_PTR<scalar_t>(), y.DATA_PTR<scalar_t>(),
        batch_x.DATA_PTR<int64_t>(), batch_y.DATA_PTR<int64_t>(),
        dist.DATA_PTR<scalar_t>(), row.DATA_PTR<int64_t>(),
        col.DATA_PTR<int64_t>(), k, x.size(1), cosine);
  });

  auto mask = col != -1;
  return at::stack({row.masked_select(mask), col.masked_select(mask)}, 0);
}

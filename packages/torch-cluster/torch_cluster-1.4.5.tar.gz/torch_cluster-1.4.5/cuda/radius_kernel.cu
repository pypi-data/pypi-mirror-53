#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include "compat.cuh"
#include "utils.cuh"

#define THREADS 1024

template <typename scalar_t>
__global__ void
radius_kernel(const scalar_t *__restrict__ x, const scalar_t *__restrict__ y,
              const int64_t *__restrict__ batch_x,
              const int64_t *__restrict__ batch_y, int64_t *__restrict__ row,
              int64_t *__restrict__ col, scalar_t radius,
              size_t max_num_neighbors, size_t dim) {

  const ptrdiff_t batch_idx = blockIdx.x;
  const ptrdiff_t idx = threadIdx.x;

  const ptrdiff_t start_idx_x = batch_x[batch_idx];
  const ptrdiff_t end_idx_x = batch_x[batch_idx + 1];

  const ptrdiff_t start_idx_y = batch_y[batch_idx];
  const ptrdiff_t end_idx_y = batch_y[batch_idx + 1];

  for (ptrdiff_t n_y = start_idx_y + idx; n_y < end_idx_y; n_y += THREADS) {
    size_t count = 0;
    for (ptrdiff_t n_x = start_idx_x; n_x < end_idx_x; n_x++) {

      scalar_t dist = 0;
      for (ptrdiff_t d = 0; d < dim; d++) {
        dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
                (x[n_x * dim + d] - y[n_y * dim + d]);
      }
      dist = sqrt(dist);

      if (dist <= radius) {
        row[n_y * max_num_neighbors + count] = n_y;
        col[n_y * max_num_neighbors + count] = n_x;
        count++;
      }

      if (count >= max_num_neighbors) {
        break;
      }
    }
  }
}

at::Tensor radius_cuda(at::Tensor x, at::Tensor y, float radius,
                       at::Tensor batch_x, at::Tensor batch_y,
                       size_t max_num_neighbors) {
  hipSetDevice(x.get_device());
  auto batch_sizes = (int64_t *)malloc(sizeof(int64_t));
  hipMemcpy(batch_sizes, batch_x[-1].DATA_PTR<int64_t>(), sizeof(int64_t),
             hipMemcpyDeviceToHost);
  auto batch_size = batch_sizes[0] + 1;

  batch_x = degree(batch_x, batch_size);
  batch_x = at::cat({at::zeros(1, batch_x.options()), batch_x.cumsum(0)}, 0);
  batch_y = degree(batch_y, batch_size);
  batch_y = at::cat({at::zeros(1, batch_y.options()), batch_y.cumsum(0)}, 0);

  auto row = at::full(y.size(0) * max_num_neighbors, -1, batch_y.options());
  auto col = at::full(y.size(0) * max_num_neighbors, -1, batch_y.options());

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "radius_kernel", [&] {
    radius_kernel<scalar_t><<<batch_size, THREADS>>>(
        x.DATA_PTR<scalar_t>(), y.DATA_PTR<scalar_t>(),
        batch_x.DATA_PTR<int64_t>(), batch_y.DATA_PTR<int64_t>(),
        row.DATA_PTR<int64_t>(), col.DATA_PTR<int64_t>(), radius,
        max_num_neighbors, x.size(1));
  });

  auto mask = row != -1;
  return at::stack({row.masked_select(mask), col.masked_select(mask)}, 0);
}

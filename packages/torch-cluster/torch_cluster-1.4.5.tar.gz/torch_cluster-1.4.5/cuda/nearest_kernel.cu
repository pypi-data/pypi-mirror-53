#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include "compat.cuh"
#include "utils.cuh"

#define THREADS 1024

template <typename scalar_t>
__global__ void nearest_kernel(const scalar_t *__restrict__ x,
                               const scalar_t *__restrict__ y,
                               const int64_t *__restrict__ batch_x,
                               const int64_t *__restrict__ batch_y,
                               int64_t *__restrict__ out, const size_t dim) {

  const ptrdiff_t n_x = blockIdx.x;
  const ptrdiff_t batch_idx = batch_x[n_x];
  const ptrdiff_t idx = threadIdx.x;

  const ptrdiff_t start_idx = batch_y[batch_idx];
  const ptrdiff_t end_idx = batch_y[batch_idx + 1];

  __shared__ scalar_t best_dist[THREADS];
  __shared__ int64_t best_dist_idx[THREADS];

  scalar_t best = 1e38;
  ptrdiff_t best_idx = 0;
  for (ptrdiff_t n_y = start_idx + idx; n_y < end_idx; n_y += THREADS) {

    scalar_t dist = 0;
    for (ptrdiff_t d = 0; d < dim; d++) {
      dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
              (x[n_x * dim + d] - y[n_y * dim + d]);
    }

    if (dist < best) {
      best = dist;
      best_idx = n_y;
    }
  }

  best_dist[idx] = best;
  best_dist_idx[idx] = best_idx;

  for (int64_t u = 0; (1 << u) < THREADS; u++) {
    __syncthreads();
    if (idx < (THREADS >> (u + 1))) {
      int64_t idx_1 = (idx * 2) << u;
      int64_t idx_2 = (idx * 2 + 1) << u;
      if (best_dist[idx_1] > best_dist[idx_2]) {
        best_dist[idx_1] = best_dist[idx_2];
        best_dist_idx[idx_1] = best_dist_idx[idx_2];
      }
    }
  }

  __syncthreads();
  if (idx == 0) {
    out[n_x] = best_dist_idx[0];
  }
}

at::Tensor nearest_cuda(at::Tensor x, at::Tensor y, at::Tensor batch_x,
                        at::Tensor batch_y) {
  hipSetDevice(x.get_device());
  auto batch_sizes = (int64_t *)malloc(sizeof(int64_t));
  hipMemcpy(batch_sizes, batch_x[-1].DATA_PTR<int64_t>(), sizeof(int64_t),
             hipMemcpyDeviceToHost);
  auto batch_size = batch_sizes[0] + 1;

  batch_y = degree(batch_y, batch_size);
  batch_y = at::cat({at::zeros(1, batch_y.options()), batch_y.cumsum(0)}, 0);

  auto out = at::empty_like(batch_x);

  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "nearest_kernel", [&] {
    nearest_kernel<scalar_t><<<x.size(0), THREADS>>>(
        x.DATA_PTR<scalar_t>(), y.DATA_PTR<scalar_t>(),
        batch_x.DATA_PTR<int64_t>(), batch_y.DATA_PTR<int64_t>(),
        out.DATA_PTR<int64_t>(), x.size(1));
  });

  return out;
}

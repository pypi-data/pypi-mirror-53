#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, Miroslav Stoyanov
 *
 * This file is part of
 * Toolkit for Adaptive Stochastic Modeling And Non-Intrusive ApproximatioN: TASMANIAN
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
 *    and the following disclaimer in the documentation and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors may be used to endorse
 *    or promote products derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY,
 * OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
 * OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * UT-BATTELLE, LLC AND THE UNITED STATES GOVERNMENT MAKE NO REPRESENTATIONS AND DISCLAIM ALL WARRANTIES, BOTH EXPRESSED AND IMPLIED.
 * THERE ARE NO EXPRESS OR IMPLIED WARRANTIES OF MERCHANTABILITY OR FITNESS FOR A PARTICULAR PURPOSE, OR THAT THE USE OF THE SOFTWARE WILL NOT INFRINGE ANY PATENT,
 * COPYRIGHT, TRADEMARK, OR OTHER PROPRIETARY RIGHTS, OR THAT THE SOFTWARE WILL ACCOMPLISH THE INTENDED RESULTS OR THAT THE SOFTWARE OR ITS USE WILL NOT RESULT IN INJURY OR DAMAGE.
 * THE USER ASSUMES RESPONSIBILITY FOR ALL LIABILITIES, PENALTIES, FINES, CLAIMS, CAUSES OF ACTION, AND COSTS AND EXPENSES, CAUSED BY, RESULTING FROM OR ARISING OUT OF,
 * IN WHOLE OR IN PART THE USE, STORAGE OR DISPOSAL OF THE SOFTWARE.
 */

#ifndef __TASMANIAN_SPARSE_GRID_CUDA_KERNELS_CU
#define __TASMANIAN_SPARSE_GRID_CUDA_KERNELS_CU

#include "tsgAcceleratedDataStructures.hpp"
#include "tsgCudaLinearAlgebra.hpp"
#include "tsgCudaBasisEvaluations.hpp"

// several kernels assume a linear distribution of the threads and can be executed with "practically unlimited" number of threads
// thus we can set this to the CUDA max number of threads, based on the current cuda version
constexpr int _MAX_CUDA_THREADS  = 1024;

namespace TasGrid{

void TasCUDA::dtrans2can(bool use01, int dims, int num_x, int pad_size, const double *gpu_trans_a, const double *gpu_trans_b, const double *gpu_x_transformed, double *gpu_x_canonical){
    int num_blocks = (num_x * dims) / _MAX_CUDA_THREADS + (((num_x * dims) % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    tasgpu_transformed_to_canonical<double, double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS, (2*pad_size) * sizeof(double)>>>(dims, num_x, pad_size, gpu_trans_a, gpu_trans_b, gpu_x_transformed, gpu_x_canonical);
    if (use01) tasgpu_m11_to_01<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>(dims * num_x, gpu_x_canonical);
}

// local polynomial basis functions, DENSE algorithm
void TasCUDA::devalpwpoly(int order, TypeOneDRule rule, int dims, int num_x, int num_points, const double *gpu_x, const double *gpu_nodes, const double *gpu_support, double *gpu_y){
    // each block thread runs 1024 threads and processes 32 points (or basis functions)
    int num_blocks = (num_points / 32) + ((num_points % 32 == 0) ? 0 : 1);
    // order == 1 is considered "default" so that the compiler doesn't complain about missing default statement
    // semilocalp cannot have order less than 2, only rule_localp can have order 0 (this gets overwrittein in makeLocalPolynomialGrid())
    if (rule == rule_localp){
        switch(order){
            case 0:
                    tasgpu_devalpwpoly<double, 0, rule_localp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            case 2: tasgpu_devalpwpoly<double, 2, rule_localp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            default:
                    tasgpu_devalpwpoly<double, 1, rule_localp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
        }
    }else if (rule == rule_localp0){
        switch(order){
            case 2: tasgpu_devalpwpoly<double, 2, rule_localp0, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            default:
                    tasgpu_devalpwpoly<double, 1, rule_localp0, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
        }
    }else if (rule == rule_localpb){
        switch(order){
            case 2: tasgpu_devalpwpoly<double, 2, rule_localpb, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            default:
                    tasgpu_devalpwpoly<double, 1, rule_localpb, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
        }
    }else if (rule == rule_semilocalp){
        tasgpu_devalpwpoly<double, 2, rule_semilocalp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
    }else{ // rule == wavelet
        tasgpu_devalpwpoly<double, 1, rule_wavelet, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
    }
}

// there is a switch statement that realizes templates for each combination of rule/order
// make one function that covers that switch, the rest is passed from devalpwpoly_sparse
template<typename T, int THREADS, int TOPLEVEL, bool fill>
inline void devalpwpoly_sparse_realize_rule_order(int order, TypeOneDRule rule,
                                          int dims, int num_x, int num_points,
                                          const T *x, const T *nodes, const T *support,
                                          const int *hpntr, const int *hindx, int num_roots, const int *roots,
                                          int *spntr, int *sindx, T *svals){
    int num_blocks = num_x / THREADS + ((num_x % THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    if (rule == rule_localp){
        switch(order){
            case 0:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 0, rule_localp, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            case 2:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_localp, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            default:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 1, rule_localp, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
        }
    }else if (rule == rule_localp0){
        switch(order){
            case 2:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_localp0, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            default:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 1, rule_localp0, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
        }
    }else if (rule == rule_localpb){
        switch(order){
            case 2:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_localpb, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            default:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 1, rule_localpb, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
        }
    }else{ // rule == rule_semilocalp
        tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_semilocalp, fill><<<num_blocks, THREADS>>>
            (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
    }
}

// local polynomial basis functions, SPARSE algorithm (2 passes, one pass to compue the non-zeros and one pass to evaluate)
void TasCUDA::devalpwpoly_sparse(int order, TypeOneDRule rule, int dims, int num_x, int num_points, const double *gpu_x,
                                 const CudaVector<double> &gpu_nodes, const CudaVector<double> &gpu_support,
                                 const CudaVector<int> &gpu_hpntr, const CudaVector<int> &gpu_hindx, const CudaVector<int> &gpu_hroots,
                                 CudaVector<int> &gpu_spntr, CudaVector<int> &gpu_sindx, CudaVector<double> &gpu_svals){
    gpu_spntr.resize(num_x + 1);
    // call with fill == false to count the non-zeros per row of the matrix
    devalpwpoly_sparse_realize_rule_order<double, 64, 46, false>
        (order, rule, dims, num_x, num_points, gpu_x, gpu_nodes.data(), gpu_support.data(),
        gpu_hpntr.data(), gpu_hindx.data(), (int) gpu_hroots.size(), gpu_hroots.data(), gpu_spntr.data(), 0, 0);

    std::vector<int> cpu_spntr;
    gpu_spntr.unload(cpu_spntr);
    cpu_spntr[0] = 0;
    int nz = 0;
    for(auto &i : cpu_spntr){
        i += nz;
        nz = i;
    }
    gpu_spntr.load(cpu_spntr);
    gpu_sindx.resize(nz);
    gpu_svals.resize(nz);
    // call with fill == true to load the non-zeros
    devalpwpoly_sparse_realize_rule_order<double, 64, 46, true>
        (order, rule, dims, num_x, num_points, gpu_x, gpu_nodes.data(), gpu_support.data(),
        gpu_hpntr.data(), gpu_hindx.data(), (int) gpu_hroots.size(), gpu_hroots.data(), gpu_spntr.data(), gpu_sindx.data(), gpu_svals.data());
}

// Sequence Grid basis evaluations
void TasCUDA::devalseq(int dims, int num_x, const std::vector<int> &max_levels, const double *gpu_x, const CudaVector<int> &num_nodes,
                       const CudaVector<int> &points, const CudaVector<double> &nodes, const CudaVector<double> &coeffs, double *gpu_result){
    std::vector<int> offsets(dims);
    offsets[0] = 0;
    for(int d=1; d<dims; d++) offsets[d] = offsets[d-1] + num_x * (max_levels[d-1] + 1);
    size_t num_total = offsets[dims-1] + num_x * (max_levels[dims-1] + 1);

    int maxl = max_levels[0]; for(auto l : max_levels) if (maxl < l) maxl = l;

    CudaVector<int> gpu_offsets(offsets);
    CudaVector<double> cache1D(num_total);
    int num_blocks = num_x / _MAX_CUDA_THREADS + ((num_x % _MAX_CUDA_THREADS == 0) ? 0 : 1);

    tasgpu_dseq_build_cache<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>
        (dims, num_x, gpu_x, nodes.data(), coeffs.data(), maxl+1, gpu_offsets.data(), num_nodes.data(), cache1D.data());

    num_blocks = num_x / 32 + ((num_x % 32 == 0) ? 0 : 1);
    tasgpu_dseq_eval_sharedpoints<double, 32><<<num_blocks, 1024>>>
        (dims, num_x, (int) points.size() / dims, points.data(), gpu_offsets.data(), cache1D.data(), gpu_result);
}

// Fourier Grid basis evaluations
void TasCUDA::devalfor(int dims, int num_x, const std::vector<int> &max_levels, const double *gpu_x, const CudaVector<int> &num_nodes, const CudaVector<int> &points, double *gpu_wreal, double *gpu_wimag){
    std::vector<int> max_nodes(dims);
    for(int j=0; j<dims; j++){
        int n = 1;
        for(int i=0; i<max_levels[j]; i++) n *= 3;
        max_nodes[j] = n;
    }

    std::vector<int> offsets(dims);
    offsets[0] = 0;
    for(int d=1; d<dims; d++) offsets[d] = offsets[d-1] + 2 * num_x * (max_nodes[d-1] + 1);
    size_t num_total = offsets[dims-1] + 2 * num_x * (max_nodes[dims-1] + 1);

    CudaVector<int> gpu_offsets(offsets);
    CudaVector<double> cache1D(num_total);
    int num_blocks = num_x / _MAX_CUDA_THREADS + ((num_x % _MAX_CUDA_THREADS == 0) ? 0 : 1);

    tasgpu_dfor_build_cache<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>
        (dims, num_x, gpu_x, gpu_offsets.data(), num_nodes.data(), cache1D.data());

    num_blocks = num_x / 32 + ((num_x % 32 == 0) ? 0 : 1);
    if (gpu_wimag == 0){
        tasgpu_dfor_eval_sharedpoints<double, 32, true><<<num_blocks, 1024>>>
            (dims, num_x, (int) points.size() / dims, points.data(), gpu_offsets.data(), cache1D.data(), gpu_wreal, 0);
    }else{
        tasgpu_dfor_eval_sharedpoints<double, 32, false><<<num_blocks, 1024>>>
            (dims, num_x, (int) points.size() / dims, points.data(), gpu_offsets.data(), cache1D.data(), gpu_wreal, gpu_wimag);
    }
}

void TasCUDA::devalglo(bool is_nested, bool is_clenshawcurtis0, int dims, int num_x, int num_p, int num_basis,
                       double const *gpu_x, CudaVector<double> const &nodes, CudaVector<double> const &coeff, CudaVector<double> const &tensor_weights,
                       CudaVector<int> const &nodes_per_level, CudaVector<int> const &offset_per_level, CudaVector<int> const &map_dimension, CudaVector<int> const &map_level,
                       CudaVector<int> const &active_tensors, CudaVector<int> const &active_num_points, CudaVector<int> const &dim_offsets,
                       CudaVector<int> const &map_tensor, CudaVector<int> const &map_index, CudaVector<int> const &map_reference, double *gpu_result){
    CudaVector<double> cache(num_x, num_basis);
    int num_blocks = (int) map_dimension.size();
    if (num_blocks >= 65536) num_blocks = 65536;

    if (is_nested){
        if (is_clenshawcurtis0){
            tasgpu_dglo_build_cache<double, _MAX_CUDA_THREADS, true, true><<<num_blocks, _MAX_CUDA_THREADS>>>
                (dims, num_x, (int) map_dimension.size(), gpu_x, nodes.data(), coeff.data(),
                                        nodes_per_level.data(), offset_per_level.data(), dim_offsets.data(),
                                        map_dimension.data(), map_level.data(), cache.data());
        }else{
            tasgpu_dglo_build_cache<double, _MAX_CUDA_THREADS, true, false><<<num_blocks, _MAX_CUDA_THREADS>>>
                (dims, num_x, (int) map_dimension.size(), gpu_x, nodes.data(), coeff.data(),
                                        nodes_per_level.data(), offset_per_level.data(), dim_offsets.data(),
                                        map_dimension.data(), map_level.data(), cache.data());
        }
    }else{
        tasgpu_dglo_build_cache<double, _MAX_CUDA_THREADS, false, false><<<num_blocks, _MAX_CUDA_THREADS>>>
            (dims, num_x, (int) map_dimension.size(), gpu_x, nodes.data(), coeff.data(),
                                    nodes_per_level.data(), offset_per_level.data(), dim_offsets.data(),
                                    map_dimension.data(), map_level.data(), cache.data());
    }

    int mat_size = num_x * num_p;
    num_blocks = num_x / _MAX_CUDA_THREADS + ((mat_size % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    tasgpu_dglo_eval_zero<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>(mat_size, gpu_result);

    num_blocks = (int) map_tensor.size();
    if (num_blocks >= 65536) num_blocks = 65536;
    tasgpu_dglo_eval_sharedpoints<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>
        (dims, num_x, (int) map_tensor.size(), num_p, cache.data(),
        tensor_weights.data(), offset_per_level.data(), dim_offsets.data(), active_tensors.data(), active_num_points.data(),
        map_tensor.data(), map_index.data(), map_reference.data(), gpu_result);

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//       Linear Algebra
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#ifdef __TASMANIAN_COMPILE_FALLBACK_CUDA_KERNELS__
void TasCUDA::cudaDgemm(int M, int N, int K, const double *gpu_a, const double *gpu_b, double *gpu_c){ // gpu_c = gpu_a * gpu_b, gpu_c is M by N
    int blocks = (N / 96) + (((N % 96) == 0) ? 0 : 1);
    blocks *= (M / 96) + (((M % 96) == 0) ? 0 : 1);
    while(blocks > 65536) blocks = 65536;
    tasgpu_cudaTgemm<double, 32, 96><<<blocks, 1024>>>(M, N, K, gpu_a, gpu_b, gpu_c);
}

void TasCUDA::cudaSparseMatmul(int M, int N, int num_nz, const int* gpu_spntr, const int* gpu_sindx, const double* gpu_svals, const double *gpu_B, double *gpu_C){
    int blocks = M / 64 + ((M % 64 == 0) ? 0 : 1);
    tasgpu_sparse_matmul<double, 64><<<blocks, 64>>>(M, N, num_nz, gpu_spntr, gpu_sindx, gpu_svals, gpu_B, gpu_C);
}

void TasCUDA::cudaSparseVecDenseMat(int M, int N, int num_nz, const double *A, const int *indx, const double *vals, double *C){
    int num_blocks = N / _MAX_CUDA_THREADS + ((N % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks< 65536){
        tasgpu_sparse_matveci<double, _MAX_CUDA_THREADS, 1><<<num_blocks, _MAX_CUDA_THREADS>>>(M, N, num_nz, A, indx, vals, C);
    }else{
        num_blocks = N / (2 * _MAX_CUDA_THREADS) + ((N % (2 * _MAX_CUDA_THREADS) == 0) ? 0 : 1);
        if (num_blocks< 65536){
            tasgpu_sparse_matveci<double, _MAX_CUDA_THREADS, 2><<<num_blocks, _MAX_CUDA_THREADS>>>(M, N, num_nz, A, indx, vals, C);
        }else{
            num_blocks = N / (3 * _MAX_CUDA_THREADS) + ((N % (3 * _MAX_CUDA_THREADS) == 0) ? 0 : 1);
            if (num_blocks< 65536){
                tasgpu_sparse_matveci<double, _MAX_CUDA_THREADS, 3><<<num_blocks, _MAX_CUDA_THREADS>>>(M, N, num_nz, A, indx, vals, C);
            }
        }
    }
}

void TasCUDA::convert_sparse_to_dense(int num_rows, int num_columns, const int *pntr, const int *indx, const double *vals, double *destination){
    int n = num_rows * num_columns;
    int num_blocks = n / _MAX_CUDA_THREADS + ((n % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    tascuda_fill<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>(n, 0.0, destination);
    num_blocks = num_rows;
    if (num_blocks >= 65536) num_blocks = 65536;
    tascuda_sparse_to_dense<double, 64><<<num_blocks, 64>>>(num_rows, num_columns, pntr, indx, vals, destination);
}
#endif

}

#endif
